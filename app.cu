#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <stdint.h>

#include "support/graph.h"
#include "support/params.h"
#include "support/timer.h"
#include "support/utils.h"


__global__ void priority_kernel(int* __restrict__ nodePtrs, 
                                int* __restrict__ neighborIdxs,
                                int* __restrict__ prioNodeIdx,
                                int* __restrict__ prioNodePtrs,
                                int numbernode) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x; //node id

    if(i < numbernode){
        const int pos = nodePtrs[i + 1] - nodePtrs[i]; //degree
        const int th_prio = pos;
        prioNodeIdx[i] = pos; //priority
        __syncthreads();

        // go through the neighbor list and get number of priority nodes
        int temp = 0;
        for(int node = nodePtrs[i]; node < nodePtrs[i + 1]; node++){
            if(prioNodeIdx[neighborIdxs[node]] > th_prio || 
                (prioNodeIdx[neighborIdxs[node]] == th_prio && neighborIdxs[node] < i)){
                temp++;
            }
        }
        prioNodePtrs[i] = temp;
    }
}

__global__ void color_kernel(int* __restrict__ nodePtrs, 
                             int* __restrict__ neighborIdxs,
                             int* __restrict__ prioNodeIdx,
                             int* __restrict__ prioNodePtrs,
                             int* __restrict__ color_num, 
                             int* __restrict__ color_bitmap,
                             int threadnum){

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < threadnum){
        int check_bit = 0;
        int pos = nodePtrs[i + 1] - nodePtrs[i]; //degree + 1

        while(1){

            //when none of its neighbors' priority is larger than itself, then color this node
            if(prioNodePtrs[i] <= 0){

                //get the first color not used by its neighbors
                for(int bit = 0; bit < pos + 1; bit++){
                    if(!isSet(color_bitmap[i], bit)){
                        check_bit = bit;
                        break;
                    }
                }
                color_num[i] = check_bit;

                //-1 to the priority of its nrighbors
                for(int node = nodePtrs[i]; node < nodePtrs[i + 1]; node++){
                    if(prioNodePtrs[neighborIdxs[node]] > 0){

                        atomicAdd(color_bitmap + neighborIdxs[node], pow(2, check_bit));
                        __threadfence();
                        atomicSub(prioNodePtrs + neighborIdxs[node], 1);
                        __ldg(&prioNodePtrs[node]);
                    
                    }
                }
                break;
            }
            __syncwarp();
        }
    }
}

int main(int argc, char** argv) {

    // Process parameters
    struct Params p = input_params(argc, argv);

    // Initialize data structures
    PRINT_INFO(p.verbosity >= 1, "Reading graph %s", p.fileName);
    struct COOGraph cooGraph = readCOOGraph(p.fileName);
    PRINT_INFO(p.verbosity >= 1, "    Graph has %d nodes and %d edges", cooGraph.numNodes, cooGraph.numEdges);
    struct CSRGraph csrGraph = coo2csr(cooGraph);
    //PRINT_INFO(p.verbosity >= 1, "    Graph has %d nodes and %d edges and %d priority edges", csrGraph.numNodes, csrGraph.numEdges, csrGraph.p_numEdges);

    //receive data from gpu
    int* color_gpu = (int*) malloc(csrGraph.numNodes*sizeof(int));
    int* prioNodeIdx_gpu = (int*) malloc (csrGraph.numNodes*sizeof(int));
    int* prioNodePtrs_gpu = (int*) malloc (csrGraph.numNodes*sizeof(int));
    int* prio_edge_gpu = (int*) malloc (csrGraph.numNodes*sizeof(int));
    for(int i = 0; i < csrGraph.numNodes; ++i) {
        color_gpu[i] = INT_MAX; // Unreachable
    }

    // Allocate GPU memory
    CSRGraph csrGraph_d;
    csrGraph_d.numNodes = csrGraph.numNodes;
    csrGraph_d.numEdges = csrGraph.numEdges;
    hipMalloc((void**) &csrGraph_d.nodePtrs, (csrGraph_d.numNodes + 1)*sizeof(int));
    hipMalloc((void**) &csrGraph_d.neighborIdxs, csrGraph_d.numEdges*sizeof(int));
    hipMalloc((void**) &csrGraph_d.prioNodePtrs, (csrGraph_d.numNodes)*sizeof(int)); //store number of neighbors with larger priority 
    hipMalloc((void**) &csrGraph_d.prioNodeIdx, csrGraph_d.numNodes*sizeof(int)); //store the random/degree based priority
    // hipMalloc((void**) &csrGraph_d.prioNeighbor, csrGraph_d.numEdges*sizeof(int)); //store larger priority node's info
    int* color_num_d;
    hipMalloc((void**) &color_num_d, csrGraph_d.numNodes*sizeof(int)); //color info for each node
    
    int* color_bitmap_d;
    hipMalloc((void**) &color_bitmap_d, csrGraph_d.numNodes*sizeof(uint32_t)); //color bitmap for each node

    // Copy data to GPU
    hipMemcpy(csrGraph_d.nodePtrs, csrGraph.nodePtrs, (csrGraph_d.numNodes + 1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(csrGraph_d.neighborIdxs, csrGraph.neighborIdxs, csrGraph_d.numEdges*sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // Calculating result on GPU
    PRINT_INFO(p.verbosity >= 1, "Calculating result on GPU");

    int numThreadsPerBlock = 256;
    int numBlocks = ceil ((double)csrGraph.numNodes / (double)numThreadsPerBlock);
    
    //kernel to give priority
    priority_kernel <<< numBlocks, numThreadsPerBlock >>> (csrGraph_d.nodePtrs, csrGraph_d.neighborIdxs, csrGraph_d.prioNodeIdx, csrGraph_d.prioNodePtrs, csrGraph_d.numNodes );
    hipDeviceSynchronize();
    
    //get data from gpu 
    // hipMemcpy(prioNodeIdx_gpu, csrGraph_d.prioNodeIdx, csrGraph_d.numNodes*sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(prioNodePtrs_gpu, csrGraph_d.prioNodePtrs, csrGraph_d.numNodes*sizeof(int), hipMemcpyDeviceToHost);

    Timer timer;
    startTimer(&timer);

    //kernel to give colors
    color_kernel <<< numBlocks, numThreadsPerBlock >>> (csrGraph_d.nodePtrs, csrGraph_d.neighborIdxs, csrGraph_d.prioNodeIdx, csrGraph_d.prioNodePtrs, color_num_d, color_bitmap_d, csrGraph_d.numNodes);
    hipDeviceSynchronize();

    stopTimer(&timer);
    if(p.verbosity == 0) PRINT("%f", getElapsedTime(timer)*1e3);
    PRINT_INFO(p.verbosity >= 1, "Elapsed time: %f ms", getElapsedTime(timer)*1e3);

    // Copy data from GPU
    hipMemcpy(color_gpu, color_num_d, csrGraph_d.numNodes*sizeof(int), hipMemcpyDeviceToHost);

    printf("color is: \n");
    for(int i = 0; i < csrGraph_d.numNodes; i++){
        printf("%d ", color_gpu[i]);
    }
    printf("\n");

    free(color_gpu);
    free(prio_edge_gpu);
    free(prioNodeIdx_gpu);
    free(prioNodePtrs_gpu);
    freeCOOGraph(cooGraph);
    freeCSRGraph(csrGraph);
    return 0;
}


